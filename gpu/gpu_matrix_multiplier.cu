#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "gpu_matrix_multiplier.h"

template<class T>
struct shared_memory
{
  __device__ inline operator T *()
  {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const
  {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

template<>
struct shared_memory<double>
{
  __device__ inline operator double *()
  {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const
  {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

#define FULL_WARP_MASK 0xFFFFFFFF

template <class T>
__device__ T warp_reduce (T val)
{
  /**
   *  For a thread at lane X in the warp, __shfl_down_sync(FULL_MASK, val, offset) gets
   *  the value of the val variable from the thread at lane X+offset of the same warp.
   *  The data exchange is performed between registers, and more efficient than going
   *  through shared memory, which requires a load, a store and an extra register to
   *  hold the address.
   */
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync (FULL_WARP_MASK, val, offset);

  return val;
}

template <typename data_type>
__global__ void fill_vector (unsigned int n, data_type *vec, data_type value)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    vec[i] = value;
}

template <typename data_type, typename index_type>
__global__ void csr_spmv_kernel (
  index_type n_rows,
  const index_type *col_ids,
  const index_type *row_ptr,
  const data_type *data,
  const data_type *x,
  data_type *y)
{
  index_type row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n_rows)
    {
      const index_type row_start = row_ptr[row];
      const index_type row_end = row_ptr[row + 1];

      data_type sum = 0;
      for (index_type element = row_start; element < row_end; element++)
        sum += data[element] * x[col_ids[element]];
      y[row] = sum;
    }
}

template <typename data_type, typename index_type>
measurement_class gpu_csr_spmv (
  const csr_matrix_class<data_type, index_type> &matrix,
  const data_type *reference_y)
{
  const index_type matrix_size = matrix.nnz;
  const index_type columns_size = matrix_size;
  const index_type row_ptr_size = matrix.n_rows + 1;
  const index_type x_size = matrix.n_cols;
  const index_type y_size = matrix.n_rows;

  data_type *d_values {};
  data_type *d_y {};
  data_type *d_x {};

  index_type *d_row_ptr {};
  index_type *d_columns {};

  hipMalloc (&d_values, matrix_size * sizeof (data_type));
  hipMalloc (&d_x, x_size * sizeof (data_type));
  hipMalloc (&d_y, y_size * sizeof (data_type));

  hipMalloc (&d_row_ptr, row_ptr_size * sizeof (index_type));
  hipMalloc (&d_columns, columns_size * sizeof (index_type));

  hipMemcpy (d_values, matrix.values.get (), matrix_size * sizeof (data_type), hipMemcpyHostToDevice);
  hipMemcpy (d_columns, matrix.columns.get (), columns_size * sizeof (index_type), hipMemcpyHostToDevice);
  hipMemcpy (d_row_ptr, matrix.row_ptr.get (), row_ptr_size * sizeof (index_type), hipMemcpyHostToDevice);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (x_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (x_size, d_x, 1.0);
  }

  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  hipDeviceSynchronize ();
  hipEventRecord (start);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (matrix.n_rows + block_size.x - 1) / block_size.x;

    csr_spmv_kernel<data_type, index_type> <<<grid_size, block_size>>> (matrix.n_rows, d_columns, d_row_ptr, d_values, d_x, d_y);
  }

  hipEventRecord (stop);
  hipEventSynchronize (stop);

  float milliseconds = 0;
  hipEventElapsedTime (&milliseconds, start, stop);
  const double elapsed = milliseconds / 1000;

  hipEventDestroy (start);
  hipEventDestroy (stop);

  std::unique_ptr<data_type[]> cpu_y (new data_type[y_size]);
  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);

  compare_results (y_size, reference_y, cpu_y.get ());

  hipFree (d_values);
  hipFree (d_x);
  hipFree (d_y);
  hipFree (d_row_ptr);
  hipFree (d_columns);

  return measurement_class ("GPU CSR", elapsed, 0, 0);
}

template <typename data_type, typename index_type>
__global__ void csr_spmv_vector_kernel (
  index_type n_rows,
  const index_type * __restrict__ col_ids,
  const index_type * __restrict__ row_ptr,
  const data_type * __restrict__ data,
  const data_type * __restrict__ x,
  data_type * __restrict__ y)
{
  const index_type thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const index_type warp_id = thread_id / 32;
  const index_type lane = thread_id % 32;

  const index_type row = warp_id; ///< One warp per row

  data_type dot = 0;
  if (row < n_rows)
    {
      const index_type row_start = row_ptr[row];
      const index_type row_end = row_ptr[row + 1];

      for (index_type element = row_start + lane; element < row_end; element += 32)
        dot += data[element] * x[col_ids[element]];
    }

  dot = warp_reduce (dot);

  if (lane == 0 && row < n_rows)
    {
      y[row] = dot;
    }
}

template <typename data_type, typename index_type>
measurement_class gpu_csr_vector_spmv (
  const csr_matrix_class<data_type, index_type> &matrix,
  const data_type *reference_y)
{
  const index_type matrix_size = matrix.nnz;
  const index_type columns_size = matrix_size;
  const index_type row_ptr_size = matrix.n_rows + 1;
  const index_type x_size = matrix.n_cols;
  const index_type y_size = matrix.n_rows;

  data_type *d_values {};
  data_type *d_y {};
  data_type *d_x {};

  index_type *d_row_ptr {};
  index_type *d_columns {};

  hipMalloc (&d_values, matrix_size * sizeof (data_type));
  hipMalloc (&d_x, x_size * sizeof (data_type));
  hipMalloc (&d_y, y_size * sizeof (data_type));

  hipMalloc (&d_row_ptr, row_ptr_size * sizeof (index_type));
  hipMalloc (&d_columns, columns_size * sizeof (index_type));

  hipMemcpy (d_values, matrix.values.get (), matrix_size * sizeof (data_type), hipMemcpyHostToDevice);
  hipMemcpy (d_columns, matrix.columns.get (), columns_size * sizeof (index_type), hipMemcpyHostToDevice);
  hipMemcpy (d_row_ptr, matrix.row_ptr.get (), row_ptr_size * sizeof (index_type), hipMemcpyHostToDevice);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (x_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (x_size, d_x, 1.0);
  }

  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  hipDeviceSynchronize ();
  hipEventRecord (start);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (matrix.n_rows * 32 + block_size.x - 1) / block_size.x;

    csr_spmv_vector_kernel<data_type, index_type> <<<grid_size, block_size>>> (matrix.n_rows, d_columns, d_row_ptr, d_values, d_x, d_y);
  }

  hipEventRecord (stop);
  hipEventSynchronize (stop);

  float milliseconds = 0;
  hipEventElapsedTime (&milliseconds, start, stop);
  const double elapsed = milliseconds / 1000;

  hipEventDestroy (start);
  hipEventDestroy (stop);

  std::unique_ptr<data_type[]> cpu_y (new data_type[y_size]);
  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);

  compare_results (y_size, reference_y, cpu_y.get ());

  hipFree (d_values);
  hipFree (d_x);
  hipFree (d_y);
  hipFree (d_row_ptr);
  hipFree (d_columns);

  return measurement_class ("GPU CSR-Vector", elapsed, 0, 0);
}

template <typename data_type, typename index_type>
__global__ void bcsr_spmv_kernel_block_per_block_row_thread_per_row_row_major_matrix (
  index_type bs,
  const index_type * __restrict__ col_ids,
  const index_type * __restrict__ row_ptr,
  const data_type * __restrict__ data,
  const data_type * __restrict__ x,
  data_type *y)
{
  const index_type row = threadIdx.x;
  const index_type block_row = blockIdx.x;
  const index_type first_block = row_ptr[block_row];
  const index_type last_block = row_ptr[block_row + 1];

  if (row < bs)
    {
      data_type local_out = 0.0;

      for (index_type block = first_block; block < last_block; block++)
        for (index_type col = 0; col < bs; col++)
          local_out += x[col_ids[block] * bs + col] * data[block * bs * bs + row * bs + col];

      y[block_row * bs + row] = local_out;
    }
}

template <typename data_type, typename index_type>
__global__ void bcsr_spmv_kernel_block_per_block_row_thread_per_row_column_major_matrix (
  index_type bs,
  const index_type * __restrict__ col_ids,
  const index_type * __restrict__ row_ptr,
  const data_type * __restrict__ data,
  const data_type * __restrict__ x,
  data_type * __restrict__ y)
{
  const index_type row = threadIdx.x;
  const index_type block_row = blockIdx.x;
  const index_type first_block = row_ptr[block_row];
  const index_type last_block = row_ptr[block_row + 1];

  if (row < bs)
    {
      data_type local_out = 0.0;

      for (index_type block = first_block; block < last_block; block++)
        for (index_type col = 0; col < bs; col++)
          local_out += x[col_ids[block] * bs + col] * data[block * bs * bs + col * bs + row];

      y[block_row * bs + row] = local_out;
    }
}

template <typename data_type, typename index_type>
__global__ void bcsr_spmv_kernel_block_per_block_row_thread_per_row_column_major_matrix_coal_x (
  index_type bs,
  const index_type * __restrict__ col_ids,
  const index_type * __restrict__ row_ptr,
  const data_type * __restrict__ data,
  const data_type * __restrict__ x,
  data_type *y)
{
  const index_type row = threadIdx.x;
  const index_type block_row = blockIdx.x;
  const index_type first_block = row_ptr[block_row];
  const index_type last_block = row_ptr[block_row + 1];

  data_type *cache_x = shared_memory<data_type> ();

  cache_x[threadIdx.x] = 0.0;
  data_type local_out = 0.0;

  for (index_type block = first_block; block < last_block; block++)
    {
      __syncthreads ();
      if (threadIdx.x < bs)
        cache_x[threadIdx.x] = x[col_ids[block] * bs + threadIdx.x];
      __syncthreads ();

      for (index_type col = 0; col < bs; col++)
        local_out += cache_x[col] * data[block * bs * bs + col * bs + row];
    }

  y[block_row * bs + row] = local_out;
}

void cusparse_bsrmv (
  hipsparseHandle_t  &handle,
  hipsparseMatDescr_t  &descr_A,
  hipsparseDirection_t direction,

  int n_rows,
  int n_cols,
  int nnzb,
  int bs,

  const float *A,
  const int *row_ptr,
  const int *columns,
  const float *x,
  float *y
  )
{
  const float alpha = 1.0;
  const float beta = 0.0;

  hipsparseSbsrmv (
    handle,
    direction,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    n_rows, n_cols, nnzb,
    &alpha, descr_A, A,
    row_ptr, columns, bs,
    x, &beta, y);
}

void cusparse_bsrmv (
  hipsparseHandle_t  &handle,
  hipsparseMatDescr_t  &descr_A,
  hipsparseDirection_t direction,

  int n_rows,
  int n_cols,
  int nnzb,
  int bs,

  const double *A,
  const int *row_ptr,
  const int *columns,
  const double *x,
  double *y
)
{
  const double alpha = 1.0;
  const double beta = 0.0;

  hipsparseDbsrmv (
    handle,
    direction,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    n_rows, n_cols, nnzb,
    &alpha, descr_A, A,
    row_ptr, columns, bs,
    x, &beta, y);
}

template <typename data_type, typename index_type>
std::vector<measurement_class> gpu_bcsr_spmv (
  bcsr_matrix_class<data_type, index_type> &matrix,
  const data_type *reference_y)
{
  std::vector<measurement_class> results;

  const index_type matrix_size = matrix.nnzb * matrix.bs * matrix.bs;
  const index_type columns_size = matrix.nnzb;
  const index_type row_ptr_size = matrix.n_rows + 1;
  const index_type x_size = matrix.n_cols * matrix.bs;
  const index_type y_size = matrix.n_rows * matrix.bs;

  data_type *d_values {};
  data_type *d_y {};
  data_type *d_x {};

  index_type *d_row_ptr {};
  index_type *d_columns {};

  hipMalloc (&d_values, matrix_size * sizeof (data_type));
  hipMalloc (&d_x, x_size * sizeof (data_type));
  hipMalloc (&d_y, y_size * sizeof (data_type));

  hipMalloc (&d_row_ptr, row_ptr_size * sizeof (index_type));
  hipMalloc (&d_columns, columns_size * sizeof (index_type));

  hipMemcpy (d_values, matrix.values.get (), matrix_size * sizeof (data_type), hipMemcpyHostToDevice);
  hipMemcpy (d_columns, matrix.columns.get (), columns_size * sizeof (index_type), hipMemcpyHostToDevice);
  hipMemcpy (d_row_ptr, matrix.row_ptr.get (), row_ptr_size * sizeof (index_type), hipMemcpyHostToDevice);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (x_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (x_size, d_x, 1.0);
  }

  {
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    hipDeviceSynchronize ();
    hipEventRecord (start);

    {
      dim3 block_size = dim3 (matrix.bs);
      dim3 grid_size {};

      grid_size.x = (matrix.n_rows * matrix.bs + block_size.x - 1) / block_size.x;

      bcsr_spmv_kernel_block_per_block_row_thread_per_row_row_major_matrix<data_type, index_type> <<<grid_size, block_size>>> (
        matrix.bs, d_columns, d_row_ptr, d_values, d_x, d_y);
    }

    hipEventRecord (stop);
    hipEventSynchronize (stop);

    float milliseconds = 0;
    hipEventElapsedTime (&milliseconds, start, stop);
    const double elapsed = milliseconds / 1000;

    hipEventDestroy (start);
    hipEventDestroy (stop);

    results.emplace_back ("GPU BCSR (row major, block per block row, thread per row)", elapsed, 0, 0);
  }

  std::unique_ptr<data_type[]> cpu_y (new data_type[y_size]);
  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);

  compare_results (y_size, reference_y, cpu_y.get ());

  /// cuSPARSE Row major
  {
    hipsparseHandle_t handle;
    hipsparseCreate (&handle);

    hipsparseMatDescr_t descr_A;
    hipsparseCreateMatDescr (&descr_A);
    hipsparseSetMatType (descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase (descr_A, HIPSPARSE_INDEX_BASE_ZERO);

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    hipDeviceSynchronize ();
    hipEventRecord (start);

    cusparse_bsrmv (handle, descr_A, HIPSPARSE_DIRECTION_ROW, matrix.n_rows, matrix.n_cols, matrix.nnzb, matrix.bs, d_values, d_row_ptr, d_columns, d_x, d_y);

    hipEventRecord (stop);
    hipEventSynchronize (stop);

    float milliseconds = 0;
    hipEventElapsedTime (&milliseconds, start, stop);
    const double elapsed = milliseconds / 1000;

    hipEventDestroy (start);
    hipEventDestroy (stop);

    hipsparseDestroyMatDescr (descr_A);
    hipsparseDestroy (handle);

    results.emplace_back ("GPU BSR (cuSPARSE, row major)", elapsed, 0, 0);

    hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);
    compare_results (y_size, reference_y, cpu_y.get ());
  }

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (y_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (y_size, d_y, 1.0);
  }

  matrix.transpose_blocks ();
  hipMemcpy (d_values, matrix.values.get (), matrix_size * sizeof (data_type), hipMemcpyHostToDevice);

  /// cuSPARSE Column major
  {
    hipsparseHandle_t handle;
    hipsparseCreate (&handle);

    hipsparseMatDescr_t descr_A;
    hipsparseCreateMatDescr (&descr_A);
    hipsparseSetMatType (descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase (descr_A, HIPSPARSE_INDEX_BASE_ZERO);

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    hipDeviceSynchronize ();
    hipEventRecord (start);

    cusparse_bsrmv (handle, descr_A, HIPSPARSE_DIRECTION_COLUMN, matrix.n_rows, matrix.n_cols, matrix.nnzb, matrix.bs, d_values, d_row_ptr, d_columns, d_x, d_y);

    hipEventRecord (stop);
    hipEventSynchronize (stop);

    float milliseconds = 0;
    hipEventElapsedTime (&milliseconds, start, stop);
    const double elapsed = milliseconds / 1000;

    hipEventDestroy (start);
    hipEventDestroy (stop);

    hipsparseDestroyMatDescr (descr_A);
    hipsparseDestroy (handle);

    results.emplace_back ("GPU BSR (cuSPARSE, column major)", elapsed, 0, 0);

    hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);
    compare_results (y_size, reference_y, cpu_y.get ());
  }

  {
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    hipDeviceSynchronize ();
    hipEventRecord (start);

    {
      dim3 block_size = dim3 (matrix.bs);
      dim3 grid_size {};

      grid_size.x = (matrix.n_rows * matrix.bs + block_size.x - 1) / block_size.x;

      bcsr_spmv_kernel_block_per_block_row_thread_per_row_column_major_matrix<data_type, index_type> <<<grid_size, block_size>>> (
        matrix.bs, d_columns, d_row_ptr, d_values, d_x, d_y);
    }

    hipEventRecord (stop);
    hipEventSynchronize (stop);

    float milliseconds = 0;
    hipEventElapsedTime (&milliseconds, start, stop);
    const double elapsed = milliseconds / 1000;

    hipEventDestroy (start);
    hipEventDestroy (stop);

    results.emplace_back ("GPU BCSR (column major, block per block row, thread per row)", elapsed, 0, 0);
  }

  std::fill_n (cpu_y.get (), y_size, 0.0);
  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);
  compare_results (y_size, reference_y, cpu_y.get ());

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (y_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (y_size, d_y, 1.0);
  }

  {
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    hipDeviceSynchronize ();
    hipEventRecord (start);

    {
      dim3 block_size = dim3 (matrix.bs);
      dim3 grid_size {};

      grid_size.x = (matrix.n_rows * matrix.bs  + block_size.x - 1) / block_size.x;

      bcsr_spmv_kernel_block_per_block_row_thread_per_row_column_major_matrix_coal_x<data_type, index_type> <<<grid_size, block_size, block_size.x * sizeof (data_type)>>> (
        matrix.bs, d_columns, d_row_ptr, d_values, d_x, d_y);
    }

    hipEventRecord (stop);
    hipEventSynchronize (stop);

    float milliseconds = 0;
    hipEventElapsedTime (&milliseconds, start, stop);
    const double elapsed = milliseconds / 1000;

    hipEventDestroy (start);
    hipEventDestroy (stop);

    results.emplace_back ("GPU BCSR (column major, block per block row, thread per row, coal x)", elapsed, 0, 0);
  }

  std::fill_n (cpu_y.get (), y_size, 0.0);
  hipMemcpy (cpu_y.get (), d_y, y_size * sizeof (data_type), hipMemcpyDeviceToHost);
  compare_results (y_size, reference_y, cpu_y.get ());

  hipFree (d_values);
  hipFree (d_x);
  hipFree (d_y);
  hipFree (d_row_ptr);
  hipFree (d_columns);

  return results;
}

#define INSTANTIATE(DTYPE,ITYPE) \
  template measurement_class gpu_csr_spmv (const csr_matrix_class<DTYPE, ITYPE> &matrix, const DTYPE *reference_y); \
  template measurement_class gpu_csr_vector_spmv (const csr_matrix_class<DTYPE, ITYPE> &matrix, const DTYPE *reference_y); \
  template std::vector<measurement_class> gpu_bcsr_spmv (bcsr_matrix_class<DTYPE, ITYPE> &matrix, const DTYPE *reference_y);

INSTANTIATE (float,int)

#undef INSTANTIATE
